#include "hip/hip_runtime.h"
/*
 * CUDA blur
 */
#ifndef _GRAV_CUDA_POTENTIAL_C_
	#define _GRAV_CUDA_POTENTIAL_C_
#endif

#include "grav_cuda.cuh"
#include ""
#include "cuda_header.cuh"

#include "grav_cpu.hpp"
#include "cuda_calls_helper.h"

#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <math.h>
using namespace std;


// variables local to this file
float* dev_coeff;
float* dev_potential;


void cuda_cpy_input_data1(){
    // GPU Coefficient file
    CUDA_CALL(hipMalloc((void**) &dev_coeff, sizeof(float) * 2*N_coeff));
    CUDA_CALL(hipMemcpy(dev_coeff, coeff, sizeof(float) * 2*N_coeff, hipMemcpyHostToDevice));

    // Vertices
    CUDA_CALL(hipMalloc((void**) &dev_vertices, sizeof(vertex) * vertices_length));
    CUDA_CALL(hipMemcpy(dev_vertices, vertices, sizeof(vertex) * vertices_length, hipMemcpyHostToDevice));

    // OUTput potential - to be compared with CPU values
    CUDA_CALL(hipMalloc((void**) &dev_potential, sizeof(float) * vertices_length));
    CUDA_CALL(hipMemset(dev_potential, 0, vertices_length* sizeof(float)));
    gpu_out_potential = (float*) malloc(sizeof(float) * vertices_length);

}

void cuda_cpy_output_data1(){
    CUDA_CALL(hipMemcpy(gpu_out_potential, dev_potential, vertices_length*sizeof(float), hipMemcpyDeviceToHost));
}

void free_gpu_memory1(){
    CUDA_CALL(hipFree(dev_coeff));
    CUDA_CALL(hipFree(dev_potential));
    CUDA_CALL(hipFree(dev_vertices));
    free(gpu_out_potential);
}


__device__ void naive_gpu_spherical_harmonics(float radius, const int n_sph, vertex dev_R_vec, float* dev_coeff, float* U, int thread_index){

    float dev_V[21*21];
    float dev_W[21*21];

    // Define pseudo coefficients
    float Radius_sq = powf(radius,2);
    float rho = powf(R_eq,2)/Radius_sq;

    float x0 = R_eq*dev_R_vec.x/Radius_sq;
    float y0 = R_eq*dev_R_vec.y/Radius_sq;
    float z0 = R_eq*dev_R_vec.z/Radius_sq;

    // Calculate zonal terms V(n, 0). Set W(n,0)=0.0
    dev_V[0]= R_eq /sqrtf(Radius_sq);
    dev_W[0] = 0.0;

    dev_V[1*(n_sph+1) + 0] = z0 *dev_V[0];
    dev_W[1*(n_sph+1) + 0] = 0.0;

    for (int n=2; n<n_sph+1; n++){
        dev_V[n*(n_sph+1) + 0] = ((2*n-1)*z0*dev_V[(n-1)*(n_sph+1) + 0] - (n-1)*rho*dev_V[(n-2)*(n_sph+1) + 0])/n;
        dev_W[n*(n_sph+1) + 0] = 0.0;
    } // Eqn 3.30


    //Calculate tesseral and sectoral terms
    for (int m = 1; m < n_sph + 1; m++){
        // Eqn 3.29
        dev_V[m*(n_sph+1) + m] = (2*m-1)*(x0*dev_V[(m-1)*(n_sph+1) + (m-1)] - y0*dev_W[(m-1)*(n_sph+1) + (m-1)]);
        dev_W[m*(n_sph+1) + m] = (2*m-1)*(x0*dev_W[(m-1)*(n_sph+1) + (m-1)] + y0*dev_V[(m-1)*(n_sph+1) + (m-1)]);

        // n=m+1 (only one term)
        if (m < n_sph){
            dev_V[(m+1)*(n_sph+1) + (m)] = (2*m+1)*z0*dev_V[m*(n_sph+1) + m];
            dev_W[(m+1)*(n_sph+1) + (m)] = (2*m+1)*z0*dev_W[m*(n_sph+1) + m] ;
        }

        for (int n = m+2; n<n_sph+1; n++){
            dev_V[n*(n_sph+1) + m] = ((2*n-1)*z0*dev_V[(n-1)*(n_sph+1) + m]-(n+m-1)*rho*dev_V[(n-2)*(n_sph+1) + m])/(n-m);
            dev_W[n*(n_sph+1) + m] = ((2*n-1)*z0*dev_W[(n-1)*(n_sph+1) + m]-(n+m-1)*rho*dev_W[(n-2)*(n_sph+1) + m])/(n-m);
        }
    }

    // Calculate potential
    float C = 0; // Cnm coeff
    float S = 0; // Snm coeff
    float N = 0; // normalisation number
    float p = 1.0;
    U[thread_index] = 0; //potential
    for (int m=0; m<n_sph+1; m++){
        for (int n = m; n<n_sph+1; n++){
            C = 0;
            S = 0;
            if (m==0){
                N = sqrtf(2*n+1);
                C = N*dev_coeff[n*(n_sph+2)+0];
//                U[thread_index] = C*dev_V[n*(n_sph+1) + 0];
            }
            else {
                p = 1.0;
                // gpu_facprod(n,m,&p);
                for (int i = n-m+1; i<=n+m; i++){
                    p = p/i;
                }
                N = sqrtf((2)*(2*n+1)*p);
                C = N*dev_coeff[n*(n_sph+2)+m];
                S = N*dev_coeff[(n_sph-n)*(n_sph+2)+ (n_sph-m+1)];
            }
            U[thread_index] = U[thread_index] + C*dev_V[n*(n_sph+1) + m] + S*dev_W[n*(n_sph+1) + m];
            // Calculation of the Gravitational Potential Calculation model
        }
    }
    U[thread_index] = U[thread_index]*mhu/R_eq;
}


__global__
void naive_kernel_gravitational(int g_vertices_length, float g_radius, const int n_sph, float* dev_coeff, vertex* dev_vertices, float* dev_potential){


    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    while (thread_index < g_vertices_length) {

        naive_gpu_spherical_harmonics(g_radius, n_sph, dev_vertices[thread_index], dev_coeff, dev_potential, thread_index);
        thread_index += blockDim.x * gridDim.x;
    }

}


void naive_cudacall_gravitational(int thread_num){

//    int n_blocks = ceil(vertices_length*1.0/thread_num);
//    n_blocks = std::min(65535, n_blocks);

    int len = vertices_length;
    int n_blocks = std::min(65535, (len + thread_num  - 1) / thread_num);
    cout<<"\n Number of blocks \t"<<n_blocks<<'\n';
    naive_kernel_gravitational<<<n_blocks, thread_num>>>(vertices_length, radius, N_SPHERICAL, dev_coeff, dev_vertices, dev_potential);
}



__global__
void optimal_kernel_gravitational(int g_vertices_length, float radius, float eq_R, const int n_sph, float* dev_coeff, vertex* dev_vertices, float* U, int* M, int* N){


//    int thread_index = (blockIdx.x * blockDim.x + threadIdx.x);
//    int potential_index = (blockIdx.x * blockDim.x + threadIdx.x)%231;

    int potential_index = blockIdx.x;

    float dev_V[21*21];
    float dev_W[21*21];

    // Define pseudo coefficients
    float Radius_sq = powf(radius,2);
    float rho = powf(eq_R,2)/Radius_sq;

    float x0 = eq_R*dev_vertices[potential_index].x/Radius_sq;
    float y0 = eq_R*dev_vertices[potential_index].y/Radius_sq;
    float z0 = eq_R*dev_vertices[potential_index].z/Radius_sq;

    // Calculate zonal terms V(n, 0). Set W(n,0)=0.0
    dev_V[0]= eq_R/radius;
    dev_W[0] = 0.0;

    dev_V[1*(n_sph+1) + 0] = z0 *dev_V[0];
    dev_W[1*(n_sph+1) + 0] = 0.0;

    for (int n=2; n<n_sph+1; n++){
        dev_V[n*(n_sph+1) + 0] = ((2*n-1)*z0*dev_V[(n-1)*(n_sph+1) + 0] - (n-1)*rho*dev_V[(n-2)*(n_sph+1) + 0])/n;
        dev_W[n*(n_sph+1) + 0] = 0.0;
    } // Eqn 3.30

    //Calculate tesseral and sectoral terms
    for (int m = 1; m < n_sph + 1; m++){
        // Eqn 3.29
        dev_V[m*(n_sph+1) + m] = (2*m-1)*(x0*dev_V[(m-1)*(n_sph+1) + (m-1)] - y0*dev_W[(m-1)*(n_sph+1) + (m-1)]);
        dev_W[m*(n_sph+1) + m] = (2*m-1)*(x0*dev_W[(m-1)*(n_sph+1) + (m-1)] + y0*dev_V[(m-1)*(n_sph+1) + (m-1)]);

        // n=m+1 (only one term)
        if (m < n_sph){
            dev_V[(m+1)*(n_sph+1) + (m)] = (2*m+1)*z0*dev_V[m*(n_sph+1) + m];
            dev_W[(m+1)*(n_sph+1) + (m)] = (2*m+1)*z0*dev_W[m*(n_sph+1) + m] ;
        }

        for (int n = m+2; n<n_sph+1; n++){
            dev_V[n*(n_sph+1) + m] = ((2*n-1)*z0*dev_V[(n-1)*(n_sph+1) + m]-(n+m-1)*rho*dev_V[(n-2)*(n_sph+1) + m])/(n-m);
            dev_W[n*(n_sph+1) + m] = ((2*n-1)*z0*dev_W[(n-1)*(n_sph+1) + m]-(n+m-1)*rho*dev_W[(n-2)*(n_sph+1) + m])/(n-m);
        }
    }

        // thread index for the block and shared memory
        unsigned int tid = threadIdx.x;

        __shared__ float shmem[256]; //stores CV+SW
        shmem[tid] = 0.0; //potential

        // Calculate potential
        float C = 0; // Cnm coeff
        float S = 0; // Snm coeff
        float Norm = 0; // normalisation number
        float p = 1.0;

        if (tid<N_coeff){
            int n = N[tid];
            int m = M[tid];

            if (m==0){
                    Norm = sqrtf(2*n+1);
                    C = Norm*dev_coeff[n*(n_sph+2)+0];
                    shmem[tid] = C*dev_V[n*(n_sph+1) + 0];
                }
                else {
                    p = 1.0;
                    for (int i = n-m+1; i<=n+m; i++){
                        p = p/i;
                    }
                    Norm = sqrtf((2)*(2*n+1)*p);
                    C = Norm*dev_coeff[n*(n_sph+2)+m];
                    S = Norm*dev_coeff[(n_sph-n)*(n_sph+2)+ (n_sph-m+1)];
                    shmem[tid] = C*dev_V[n*(n_sph+1) + m] + S*dev_W[n*(n_sph+1) + m];
                }
        }
        // Calculation of the Gravitational Potential Calculation model

        // sync threads before commencing the stages of reduction
        __syncthreads();

        // Reduction #3: Sequential Addressing
        // Ref: Presentation "Optimizing Parallel Reduction in CUDA", by Mark Harris.
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            // conduct the summation
            shmem[tid] = shmem[tid] + shmem[tid + s];
//                atomicAdd(&shmem[tid], shmem[tid + s]);
        }
        // Sync threads after every stage of reduction
        __syncthreads();
    }

//    U[potential_index] = shmem[0]*mhu/R_eq;
    U[potential_index] = shmem[0];
//        thread_index += blockDim.x * gridDim.x;
//    }

}


__global__
void optimal_kernel_gravitational2(int g_vertices_length, float radius, float eq_R, const int n_sph, float* dev_coeff, vertex* dev_vertices, float* U, int* M, int* N){


//    int thread_index = (blockIdx.x * blockDim.x + threadIdx.x);
//    int potential_index = (blockIdx.x * blockDim.x + threadIdx.x)%231;

    int potential_index = blockIdx.x;

    __shared__ float dev_V[21*21];
    __shared__ float dev_W[21*21];

    // Define pseudo coefficients
    float Radius_sq = powf(radius,2);
    float rho = powf(eq_R,2)/Radius_sq;

    float x0 = eq_R*dev_vertices[potential_index].x/Radius_sq;
    float y0 = eq_R*dev_vertices[potential_index].y/Radius_sq;
    float z0 = eq_R*dev_vertices[potential_index].z/Radius_sq;

    // Calculate zonal terms V(n, 0). Set W(n,0)=0.0
    dev_V[0]= eq_R/radius;
    dev_W[0] = 0.0;

    __syncthreads();

    dev_V[1*(n_sph+1) + 0] = z0 *dev_V[0];
    dev_W[1*(n_sph+1) + 0] = 0.0;

    __syncthreads();

    for (int n=2; n<n_sph+1; n++){
        dev_V[n*(n_sph+1) + 0] = ((2*n-1)*z0*dev_V[(n-1)*(n_sph+1) + 0] - (n-1)*rho*dev_V[(n-2)*(n_sph+1) + 0])/n;
        dev_W[n*(n_sph+1) + 0] = 0.0;
        __syncthreads();

    } // Eqn 3.30

    //Calculate tesseral and sectoral terms
    for (int m = 1; m < n_sph + 1; m++){
        // Eqn 3.29
        dev_V[m*(n_sph+1) + m] = (2*m-1)*(x0*dev_V[(m-1)*(n_sph+1) + (m-1)] - y0*dev_W[(m-1)*(n_sph+1) + (m-1)]);
        __syncthreads();

        dev_W[m*(n_sph+1) + m] = (2*m-1)*(x0*dev_W[(m-1)*(n_sph+1) + (m-1)] + y0*dev_V[(m-1)*(n_sph+1) + (m-1)]);
        __syncthreads();

        // n=m+1 (only one term)
        if (m < n_sph){
            dev_V[(m+1)*(n_sph+1) + (m)] = (2*m+1)*z0*dev_V[m*(n_sph+1) + m];
            dev_W[(m+1)*(n_sph+1) + (m)] = (2*m+1)*z0*dev_W[m*(n_sph+1) + m] ;
            __syncthreads();

        }

        for (int n = m+2; n<n_sph+1; n++){
            dev_V[n*(n_sph+1) + m] = ((2*n-1)*z0*dev_V[(n-1)*(n_sph+1) + m]-(n+m-1)*rho*dev_V[(n-2)*(n_sph+1) + m])/(n-m);
            dev_W[n*(n_sph+1) + m] = ((2*n-1)*z0*dev_W[(n-1)*(n_sph+1) + m]-(n+m-1)*rho*dev_W[(n-2)*(n_sph+1) + m])/(n-m);
            __syncthreads();

        }
    }

        // thread index for the block and shared memory
        unsigned int tid = threadIdx.x;

        __shared__ float shmem[256]; //stores CV+SW
        shmem[tid] = 0.0; //potential

        // Calculate potential
        float C = 0; // Cnm coeff
        float S = 0; // Snm coeff
        float Norm = 0; // normalisation number
        float p = 1.0;

        if (tid<N_coeff){
            int n = N[tid];
            int m = M[tid];

            if (m==0){
                    Norm = sqrtf(2*n+1);
                    C = Norm*dev_coeff[n*(n_sph+2)+0];
                    shmem[tid] = C*dev_V[n*(n_sph+1) + 0];
                }
                else {
                    p = 1.0;
                    for (int i = n-m+1; i<=n+m; i++){
                        p = p/i;
                    }
                    Norm = sqrtf((2)*(2*n+1)*p);
                    C = Norm*dev_coeff[n*(n_sph+2)+m];
                    S = Norm*dev_coeff[(n_sph-n)*(n_sph+2)+ (n_sph-m+1)];
                    shmem[tid] = C*dev_V[n*(n_sph+1) + m] + S*dev_W[n*(n_sph+1) + m];
                }
        }
        // Calculation of the Gravitational Potential Calculation model

        // sync threads before commencing the stages of reduction
        __syncthreads();

        // Reduction #3: Sequential Addressing
        // Ref: Presentation "Optimizing Parallel Reduction in CUDA", by Mark Harris.
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            // conduct the summation
            shmem[tid] = shmem[tid] + shmem[tid + s];
//                atomicAdd(&shmem[tid], shmem[tid + s]);
        }
        // Sync threads after every stage of reduction
        __syncthreads();
    }

//    U[potential_index] = shmem[0]*mhu/R_eq;
    U[potential_index] = shmem[0];
//        thread_index += blockDim.x * gridDim.x;
//    }

}




void optimal_cudacall_gravitational(int thread_num){

//    int n_blocks = ceil(vertices_length*1.0/thread_num);
//    n_blocks = std::min(65535, n_blocks);

    int len = vertices_length;
//    int n_blocks = std::min(65535, (len + thread_num  - 1) / thread_num);
    int n_blocks = std::min(65535, len);
    cout<<"\n Number of blocks \t"<<n_blocks<<'\n';

    int M[N_coeff];
    int N[N_coeff];

    int k = 0;
    for (int n=0;n<N_SPHERICAL+1;n++){
        for (int m=0;m<n+1;m++){
            N[k] = n;
            M[k] = m;
            k++;
        }
    }

    int* dev_M;
    int* dev_N;

    CUDA_CALL(hipMalloc((void**) &dev_N, sizeof(int) * N_coeff));
    CUDA_CALL(hipMalloc((void**) &dev_M, sizeof(int) * N_coeff));
    CUDA_CALL(hipMemcpy(dev_N, N, sizeof(int) * N_coeff, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(dev_M, M, sizeof(int) * N_coeff, hipMemcpyHostToDevice));
    optimal_kernel_gravitational<<<n_blocks, thread_num>>>(vertices_length, radius, R_eq, N_SPHERICAL, dev_coeff, dev_vertices, dev_potential, dev_M, dev_N);
    CUDA_CALL(hipFree(dev_M));
    CUDA_CALL(hipFree(dev_N));
}


__global__
void optimal_kernel_gravitational3(int g_vertices_length, float radius, float eq_R, const int n_sph, float* dev_coeff, vertex* dev_vertices, float* U){


    int thread_index = (blockIdx.x * blockDim.x + threadIdx.x);
    int tid = threadIdx.x;

    int vertex_index = thread_index-16*blockIdx.x;
    if(tid>=16)
        vertex_index = vertex_index-16;


    __shared__ float dev_VW[21*22*16];

    // Define pseudo coefficients
    float Radius_sq = powf(radius,2);
    float rho = powf(eq_R,2)/Radius_sq;

    float x0 = eq_R*dev_vertices[vertex_index].x/Radius_sq;
    float y0 = eq_R*dev_vertices[vertex_index].y/Radius_sq;
    float z0 = eq_R*dev_vertices[vertex_index].z/Radius_sq;

    // Calculate zonal terms V(n, 0). Set W(n,0)=0.0

    if (tid<16)
        dev_VW[tid*(n_sph+1)*(n_sph+2) + 0*(n_sph+2)+0] = eq_R/radius;
    else
        dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-0)*(n_sph+2)+ (n_sph+1-0)] = 0.0;

    __syncthreads();

    if (tid<16)
        dev_VW[tid*(n_sph+1)*(n_sph+2) + 1*(n_sph+2)+0] = z0 *eq_R/radius;
    else
        dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-1)*(n_sph+2)+ (n_sph+1-0)] = 0.0;

    __syncthreads();

    if (tid<16){
        for (int n=2; n<n_sph+1; n++){
            dev_VW[tid*(n_sph+1)*(n_sph+2) + n*(n_sph+2)+0] = ((2*n-1)*z0*dev_VW[tid*(n_sph+1)*(n_sph+2) + (n-1)*(n_sph+2)+0] - (n-1)*rho*dev_VW[tid*(n_sph+1)*(n_sph+2) + (n-2)*(n_sph+2)+0])/n;
//            __syncthreads();
        } // Eqn 3.30
    }
    else{
        for (int n=2; n<n_sph+1; n++){
            dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-n)*(n_sph+2)+ (n_sph+1-0)] = 0.0;
        } // Eqn 3.30

    }

    //Calculate tesseral and sectoral terms
    for (int m = 1; m < n_sph + 1; m++){
        // Eqn 3.29
        if(tid<16){
            dev_VW[tid*(n_sph+1)*(n_sph+2) + m*(n_sph+2)+m] = (2*m-1)*(x0*dev_VW[tid*(n_sph+1)*(n_sph+2) + (m-1)*(n_sph+2)+m-1]- y0*dev_VW[(tid)*(n_sph+1)*(n_sph+2) + (n_sph-(m-1))*(n_sph+2)+ (n_sph+1-(m-1))]);
//            __syncthreads();
        }
        else{
            dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-(m))*(n_sph+2)+ (n_sph+1-(m))] = (2*m-1)*(x0*dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-(m-1))*(n_sph+2)+ (n_sph+1-(m-1))] + y0*dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (m-1)*(n_sph+2)+m-1]);
//            __syncthreads();
        }
    }

    if(tid<16){
        for (int m = 1; m < n_sph + 1; m++){
            // n=m+1 (only one term)
            if (m < n_sph){
                dev_VW[tid*(n_sph+1)*(n_sph+2) + (m+1)*(n_sph+2)+m] = (2*m+1)*z0*dev_VW[tid*(n_sph+1)*(n_sph+2) + m*(n_sph+2)+m];

//                __syncthreads();

            }

            for (int n = m+2; n<n_sph+1; n++){
                dev_VW[tid*(n_sph+1)*(n_sph+2) + n*(n_sph+2)+m] = ((2*n-1)*z0*dev_VW[tid*(n_sph+1)*(n_sph+2) + (n-1)*(n_sph+2)+m]-(n+m-1)*rho*dev_VW[tid*(n_sph+1)*(n_sph+2) + (n-2)*(n_sph+2)+m])/(n-m);
//                __syncthreads();
            }
        }
    }
    else{
        for (int m = 1; m < n_sph + 1; m++){
            // n=m+1 (only one term)
            if (m < n_sph){
                dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-(m+1))*(n_sph+2)+ (n_sph+1-(m))] = (2*m+1)*z0*dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-(m))*(n_sph+2)+ (n_sph+1-(m))];
//                __syncthreads();

            }

            for (int n = m+2; n<n_sph+1; n++){
                dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-(n))*(n_sph+2)+ (n_sph+1-(m))] = ((2*n-1)*z0*dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-(n-1))*(n_sph+2)+ (n_sph+1-(m))]-(n+m-1)*rho*dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-(n-2))*(n_sph+2)+ (n_sph+1-(m))])/(n-m);
//                __syncthreads();
            }
        }
    }
    __syncthreads();

    __shared__ float shmem[2*16]; //stores CV+SW
    shmem[tid] = 0.0; //potential

    // Calculate potential
    float C = 0; // Cnm coeff
    float S = 0; // Snm coeff
    float N = 0; // normalisation number
    float p = 1.0;
//    U[vertex_index] = 0.0; //potential
    for (int m=0; m<n_sph+1; m++){
        for (int n = m; n<n_sph+1; n++){
//            C = 0;
            S = 0;
            if (m==0){
                N = sqrtf(2*n+1);
                C = N*dev_coeff[n*(n_sph+2)+0];
            }
            else {
                p = 1.0;
                // gpu_facprod(n,m,&p);
                for (int i = n-m+1; i<=n+m; i++){
                    p = p/i;
                }
                N = sqrtf((2)*(2*n+1)*p);
                C = N*dev_coeff[n*(n_sph+2)+m];
                S = N*dev_coeff[(n_sph-n)*(n_sph+2)+ (n_sph-m+1)];
            }
            if(tid<16){
                shmem[tid] = shmem[tid] + C*dev_VW[tid*(n_sph+1)*(n_sph+2) + n*(n_sph+2)+m];
                __syncthreads();
            }
            else{
                shmem[tid] = shmem[tid] + S*dev_VW[(tid-16)*(n_sph+1)*(n_sph+2) + (n_sph-(n))*(n_sph+2)+ (n_sph+1-(m))];
            // Calculation of the Gravitational Potential Calculation model
            __syncthreads();
            }
        }
    }

    if(tid<16){
        U[vertex_index] = shmem[tid] + shmem[tid+16];
        __syncthreads();
        U[vertex_index] = U[vertex_index]*mhu/R_eq;
    }

}


__global__
void optimal_kernel_gravitational4(int g_vertices_length, float radius, float eq_R, const int n_sph, float* dev_coeff, vertex* dev_vertices, float* U){


    int thread_index = (blockIdx.x * blockDim.x + threadIdx.x);
    int tid = threadIdx.x;
    int blk_hlf_dim = blockDim.x/2;


    int vertex_index = thread_index-blk_hlf_dim*blockIdx.x;
    if(tid>=blk_hlf_dim)
        vertex_index = vertex_index-blk_hlf_dim;


    __shared__ float dev_VW[21*22*32];

    // Define pseudo coefficients
    float Radius_sq = powf(radius,2);
    float rho = powf(eq_R,2)/Radius_sq;

    float x0 = eq_R*dev_vertices[vertex_index].x/Radius_sq;
    float y0 = eq_R*dev_vertices[vertex_index].y/Radius_sq;
    float z0 = eq_R*dev_vertices[vertex_index].z/Radius_sq;

    // Calculate zonal terms V(n, 0). Set W(n,0)=0.0

    if (tid<blk_hlf_dim)
        dev_VW[tid*(n_sph+1)*(n_sph+2) + 0*(n_sph+2)+0] = eq_R/radius;
    else
        dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-0)*(n_sph+2)+ (n_sph+1-0)] = 0.0;

    __syncthreads();

    if (tid<blk_hlf_dim)
        dev_VW[tid*(n_sph+1)*(n_sph+2) + 1*(n_sph+2)+0] = z0 *eq_R/radius;
    else
        dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-1)*(n_sph+2)+ (n_sph+1-0)] = 0.0;

    __syncthreads();

    if (tid<blk_hlf_dim){
        for (int n=2; n<n_sph+1; n++){
            dev_VW[tid*(n_sph+1)*(n_sph+2) + n*(n_sph+2)+0] = ((2*n-1)*z0*dev_VW[tid*(n_sph+1)*(n_sph+2) + (n-1)*(n_sph+2)+0] - (n-1)*rho*dev_VW[tid*(n_sph+1)*(n_sph+2) + (n-2)*(n_sph+2)+0])/n;
//            __syncthreads();
        } // Eqn 3.30
    }
    else{
        for (int n=2; n<n_sph+1; n++){
            dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-n)*(n_sph+2)+ (n_sph+1-0)] = 0.0;
        } // Eqn 3.30

    }

    //Calculate tesseral and sectoral terms
    for (int m = 1; m < n_sph + 1; m++){
        // Eqn 3.29
        if(tid<blk_hlf_dim){
            dev_VW[tid*(n_sph+1)*(n_sph+2) + m*(n_sph+2)+m] = (2*m-1)*(x0*dev_VW[tid*(n_sph+1)*(n_sph+2) + (m-1)*(n_sph+2)+m-1]- y0*dev_VW[(tid)*(n_sph+1)*(n_sph+2) + (n_sph-(m-1))*(n_sph+2)+ (n_sph+1-(m-1))]);
//            __syncthreads();
        }
        else{
            dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-(m))*(n_sph+2)+ (n_sph+1-(m))] = (2*m-1)*(x0*dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-(m-1))*(n_sph+2)+ (n_sph+1-(m-1))] + y0*dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (m-1)*(n_sph+2)+m-1]);
//            __syncthreads();
        }
    }

    if(tid<blk_hlf_dim){
        for (int m = 1; m < n_sph + 1; m++){
            // n=m+1 (only one term)
            if (m < n_sph){
                dev_VW[tid*(n_sph+1)*(n_sph+2) + (m+1)*(n_sph+2)+m] = (2*m+1)*z0*dev_VW[tid*(n_sph+1)*(n_sph+2) + m*(n_sph+2)+m];

//                __syncthreads();

            }

            for (int n = m+2; n<n_sph+1; n++){
                dev_VW[tid*(n_sph+1)*(n_sph+2) + n*(n_sph+2)+m] = ((2*n-1)*z0*dev_VW[tid*(n_sph+1)*(n_sph+2) + (n-1)*(n_sph+2)+m]-(n+m-1)*rho*dev_VW[tid*(n_sph+1)*(n_sph+2) + (n-2)*(n_sph+2)+m])/(n-m);
//                __syncthreads();
            }
        }
    }
    else{
        for (int m = 1; m < n_sph + 1; m++){
            // n=m+1 (only one term)
            if (m < n_sph){
                dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-(m+1))*(n_sph+2)+ (n_sph+1-(m))] = (2*m+1)*z0*dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-(m))*(n_sph+2)+ (n_sph+1-(m))];
//                __syncthreads();

            }

            for (int n = m+2; n<n_sph+1; n++){
                dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-(n))*(n_sph+2)+ (n_sph+1-(m))] = ((2*n-1)*z0*dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-(n-1))*(n_sph+2)+ (n_sph+1-(m))]-(n+m-1)*rho*dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-(n-2))*(n_sph+2)+ (n_sph+1-(m))])/(n-m);
//                __syncthreads();
            }
        }
    }
    __syncthreads();

    __shared__ float shmem[2*32]; //stores CV+SW
    shmem[tid] = 0.0; //potential

    // Calculate potential
    float C = 0; // Cnm coeff
    float S = 0; // Snm coeff
    float N = 0; // normalisation number
    float p = 1.0;
//    U[vertex_index] = 0.0; //potential
    for (int m=0; m<n_sph+1; m++){
        for (int n = m; n<n_sph+1; n++){
//            C = 0;
            S = 0;
            if (m==0){
                N = sqrtf(2*n+1);
                C = N*dev_coeff[n*(n_sph+2)+0];
            }
            else {
                p = 1.0;
                // gpu_facprod(n,m,&p);
                for (int i = n-m+1; i<=n+m; i++){
                    p = p/i;
                }
                N = sqrtf((2)*(2*n+1)*p);
                C = N*dev_coeff[n*(n_sph+2)+m];
                S = N*dev_coeff[(n_sph-n)*(n_sph+2)+ (n_sph-m+1)];
            }
            if(tid<blk_hlf_dim){
                shmem[tid] = shmem[tid] + C*dev_VW[tid*(n_sph+1)*(n_sph+2) + n*(n_sph+2)+m];
                __syncthreads();
            }
            else{
                shmem[tid] = shmem[tid] + S*dev_VW[(tid-blk_hlf_dim)*(n_sph+1)*(n_sph+2) + (n_sph-(n))*(n_sph+2)+ (n_sph+1-(m))];
            // Calculation of the Gravitational Potential Calculation model
            __syncthreads();
            }
        }
    }

    if(tid<blk_hlf_dim){
        U[vertex_index] = shmem[tid] + shmem[tid+blk_hlf_dim];
        __syncthreads();
        U[vertex_index] = U[vertex_index]*mhu/R_eq;
    }

}


void optimal_cudacall_gravitational3(){

    // Number of threads/ block = 32;
    // Number of vertices/block = 16;
    // Compute V, W in shared memory and separately


    int len = vertices_length;
    int n_blocks = ceil(len*1.0/16);
    n_blocks = std::min(65535,  n_blocks);
    cout<<"\n Number of blocks \t"<<n_blocks<<'\n';
    optimal_kernel_gravitational3<<<n_blocks, 32>>>(vertices_length, radius, R_eq, N_SPHERICAL, dev_coeff, dev_vertices, dev_potential);

}



void optimal_cudacall_gravitational4(){

    // Number of threads/ block = 64;
    // Number of vertices/block = 32;
    // Compute V, W in shared memory and separately

    int len = vertices_length;
    int n_blocks = ceil(len*1.0/32);
    n_blocks = std::min(65535,  n_blocks);
    cout<<"\n Number of blocks \t"<<n_blocks<<'\n';
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(optimal_kernel_gravitational4), hipFuncCachePreferShared);
    optimal_kernel_gravitational4<<<n_blocks, 64>>>(vertices_length, radius, R_eq, N_SPHERICAL, dev_coeff, dev_vertices, dev_potential);
}

