#include "hip/hip_runtime.h"
/*
 * CUDA blur
 */
#ifndef _GRAV_CUDA_C_
	#define _GRAV_CUDA_C_
#endif

#include "grav_cuda.cuh"
#include ""
#include "cuda_header.cuh"

#include "grav_cpu.hpp"
#include "cuda_calls_helper.h"

#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <math.h>
using namespace std;


// variables local to this file
float* dev_coeff;
float* dev_potential;
vertex* dev_vertices;

int ind2_faces;
triangle * pointers[2];
triangle * dev_faces_in;
triangle * dev_faces_out;

int * pointers_inds[2];
int ind2_inds;
int * dev_face_vert_ind;
int * dev_face_vert_ind_res;


float * pointers_sums[2];
int ind2_sums;
float * dev_face_sums;
float * dev_face_sums_res;

triangle * dev_verties;


void cuda_cpy_input_data(){
    gpu_out_faces = (triangle *)malloc(faces_length*sizeof(triangle));
    CUDA_CALL(hipMalloc((void **)&dev_faces_in, faces_length * sizeof(triangle)));
    CUDA_CALL(hipMalloc((void **)&dev_faces_out, faces_length * sizeof(triangle)));
    
    CUDA_CALL(hipMalloc((void **)&dev_face_vert_ind, 3*faces_length * sizeof(int)));
    CUDA_CALL(hipMalloc((void **)&dev_face_vert_ind_res, 3*faces_length * sizeof(int)));
    
    CUDA_CALL(hipMalloc((void **)&dev_face_sums, 3*faces_length * sizeof(float)));    
    CUDA_CALL(hipMalloc((void**) &dev_face_sums_res, 3*faces_length* sizeof(float)));

    CUDA_CALL(hipMemcpy(dev_faces_in, faces_init, ICOSPHERE_INIT_FACE_LEN*sizeof(triangle), hipMemcpyHostToDevice));

    ind2_faces = 0;
    pointers[0] = dev_faces_in;
    pointers[1] = dev_faces_out;

    ind2_sums = 0;
    pointers_sums[0] = dev_face_sums;
    pointers_sums[1] = dev_face_sums_res;

    ind2_inds = 0;
    pointers_inds[0] = dev_face_vert_ind;
    pointers_inds[1] = dev_face_vert_ind_res;

    // GARIMA DATA

    // GPU Coefficient file
    CUDA_CALL(hipMalloc((void**) &dev_coeff, sizeof(float) * 2*N_coeff));
    CUDA_CALL(hipMemcpy(dev_coeff, coeff, sizeof(float) * 2*N_coeff, hipMemcpyHostToDevice));

    // Vertices
    CUDA_CALL(hipMalloc((void**) &dev_vertices, sizeof(vertex) * vertices_length));
    CUDA_CALL(hipMemcpy(dev_vertices, vertices, sizeof(vertex) * vertices_length, hipMemcpyHostToDevice));

    // OUTput potential - to be compared with CPU values
    CUDA_CALL(hipMalloc((void**) &dev_potential, sizeof(float) * vertices_length));
    CUDA_CALL(hipMemset(dev_potential, 0, vertices_length* sizeof(float)));
    gpu_out_potential = (float*) malloc(sizeof(float) * vertices_length);

}

void cuda_cpy_output_data(){
    CUDA_CALL(hipMemcpy(gpu_out_faces, pointers[ind2_faces], faces_length*sizeof(triangle), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(gpu_out_potential, dev_potential, vertices_length*sizeof(float), hipMemcpyDeviceToHost));
}

void free_gpu_memory(){
    CUDA_CALL(hipFree(dev_faces_in));
    CUDA_CALL(hipFree(dev_faces_out));
    
    CUDA_CALL(hipFree(dev_face_vert_ind));
    CUDA_CALL(hipFree(dev_face_vert_ind_res));
    
    CUDA_CALL(hipFree(dev_face_sums));
    CUDA_CALL(hipFree(dev_face_sums_res));


    CUDA_CALL(hipFree(dev_coeff));
    CUDA_CALL(hipFree(dev_potential));
    CUDA_CALL(hipFree(dev_vertices));
    free(gpu_out_faces);
    free(gpu_out_potential);
}


__device__ void break_triangle(triangle face_tmp, vertex * v_tmp, float radius) {
	float x_tmp, y_tmp, z_tmp, scale;
    for(int i=0; i<3; i++){
    	x_tmp = (face_tmp.v[i].x + face_tmp.v[(i+1)%3].x)/2;
		y_tmp = (face_tmp.v[i].y + face_tmp.v[(i+1)%3].y)/2;
		z_tmp = (face_tmp.v[i].z + face_tmp.v[(i+1)%3].z)/2;
		scale = radius/sqrtf(x_tmp*x_tmp + y_tmp*y_tmp + z_tmp*z_tmp);
		v_tmp[i].x = x_tmp*scale;
		v_tmp[i].y = y_tmp*scale;
		v_tmp[i].z = z_tmp*scale;
    }
}

__global__ void refine_icosphere_naive_kernal(triangle * faces, const float radius, const unsigned int depth) {

	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int numthrds = blockDim.x * gridDim.x;

	unsigned int  write_offset;

	vertex v_tmp[3];

	const unsigned int th_len = 20*pow(4, depth);
	while(idx < th_len){

		triangle tri_tmp = faces[idx];
		write_offset = th_len + 3*idx;

		break_triangle(tri_tmp, v_tmp, radius);
		// got the mid points of the vertices now make new triangles
		faces[idx].v[1] = v_tmp[0];
		faces[idx].v[2] = v_tmp[2];

		// adding triangle V[0], P1, V[1]
		faces[write_offset].v[0] = v_tmp[0];
		faces[write_offset].v[1] = tri_tmp.v[1];
		faces[write_offset].v[2] = v_tmp[1];
		write_offset++;

		//adding triangle P2, V[1], V[2]
		faces[write_offset].v[0] = v_tmp[1];
		faces[write_offset].v[1] = tri_tmp.v[2];
		faces[write_offset].v[2] = v_tmp[2];
		write_offset++;

		//adding triangle V[0], V[1], V[2]
		faces[write_offset].v[0] = v_tmp[0];
		faces[write_offset].v[1] = v_tmp[1];
		faces[write_offset].v[2] = v_tmp[2];
		write_offset++;

		idx += numthrds;
	}

}

void cudacall_icosphere_naive(int thread_num) {
	// each thread works on one face
	for(int i=0; i<max_depth; i++){
		int ths = 20*pow(4, i);
		int n_blocks = std::min(65535, (ths + thread_num  - 1) / thread_num);
		refine_icosphere_naive_kernal<<<n_blocks, thread_num>>>(dev_faces_in, radius, i);
	}

}


typedef void (*func_ptr_sub_triangle_t)(triangle, vertex *, triangle *);

__device__ void sub_triangle_top(triangle face_tmp, vertex * v_tmp, triangle * res) {
    res->v[0] = face_tmp.v[0];
    res->v[1] = v_tmp[0];
    res->v[2] = v_tmp[2];
}

__device__ void sub_triangle_left(triangle face_tmp, vertex * v_tmp, triangle * res) {
    res->v[0] = v_tmp[0];
    res->v[1] = face_tmp.v[1];
    res->v[2] = v_tmp[1];
}

__device__ void sub_triangle_right(triangle face_tmp, vertex * v_tmp, triangle * res) {
    res->v[0] = v_tmp[1];
    res->v[1] = face_tmp.v[2];
    res->v[2] = v_tmp[2];
}

__device__ void sub_triangle_center(triangle face_tmp, vertex * v_tmp, triangle * res) {
    res->v[0] = v_tmp[0];
    res->v[1] = v_tmp[1];
    res->v[2] = v_tmp[2];
}

__device__ func_ptr_sub_triangle_t funcs2[4] = {sub_triangle_top, sub_triangle_left, sub_triangle_right, sub_triangle_center};


__global__ void refine_icosphere_kernal(triangle * faces, float * sums, const float radius, const unsigned int th_len, triangle * faces_out) {
    	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int numthrds = blockDim.x * gridDim.x;

	vertex v_tmp[3];

	while(idx < 4*th_len){
		int tri_ind = idx/4;
		int sub_tri_ind = idx%4;

		break_triangle(faces[tri_ind], v_tmp, radius);

		funcs2[sub_tri_ind](faces[tri_ind], v_tmp, &faces_out[idx]);

		idx += numthrds;
	}

}


__global__ 
void kernal_fill_sums_inds(vertex * vs, float * sums, int * inds, const unsigned int vertices_length){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numthrds = blockDim.x * gridDim.x;

    while(idx < vertices_length){
        sums[idx] = vs[idx].x + vs[idx].y + vs[idx].z;
        inds[idx] = idx;
        idx += numthrds;
    }
}

void cudacall_icosphere(int thread_num) {
	// each thread creates a sub triangle
	int ths, n_blocks, ind1;
	for(int i=0; i<max_depth; i++){
		ths = 20*pow(4, i);
		n_blocks = std::min(65535, (ths + thread_num  - 1) / thread_num);
		ind1 = i%2;
		ind2_faces = (i+1)%2;
		refine_icosphere_kernal<<<n_blocks, thread_num>>>(pointers[ind1], dev_face_sums, radius, ths, pointers[ind2_faces]);
	}
    int len = 3*faces_length;
    n_blocks = std::min(65535, (len + thread_num  - 1) / thread_num);
    kernal_fill_sums_inds<<<n_blocks, thread_num>>>((vertex *)pointers[ind2_faces], dev_face_sums, dev_face_vert_ind, len);
}


__device__
void dev_merge(float * s, float * r, int * ind, int * ind_res, unsigned int idx, unsigned int start, unsigned int end){
    unsigned int c=idx;
    unsigned int i=idx;unsigned int j=start;
    while(j<end && i<start){
        if(s[i] <= s[j]){
            r[c] = s[i];
            ind_res[c] = ind[i];
            i++;
        }
        else{
            r[c] = s[j];
            ind_res[c] = ind[j];
            j++;
        }
        c++;
    }
    while(i < start){
        r[c] = s[i];
        ind_res[c] = ind[i];
        c++;i++;
    }
    
    while(j < end){
        r[c] = s[j];
        ind_res[c] = ind[j];
        c++;j++;
    }
}

__global__
void kernal_merge_navie_sort(float * sums, float * res, int * ind, int * ind_res, const unsigned int length, const unsigned int r){
    
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numthrds = blockDim.x * gridDim.x;

    const unsigned int stride = r/2;
    
    while(idx < length){
        if(idx%r == 0)
            dev_merge(sums, res, ind, ind_res, idx, (unsigned int)min(length, idx + stride), (unsigned int)min(length, idx+r));
        idx += numthrds;
    }
}

__global__
void kernal_merge_sort(float * sums, float * res, int * ind, int * ind_res, const unsigned int length, const unsigned int r){
    
    __shared__ float sh_sums[1024];
    __shared__ float sh_res[1024];
    __shared__ int sh_ind[1024];
    __shared__ int sh_indres[1024];
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numthrds = blockDim.x * gridDim.x;

    const int stride = r/2;

    int id = threadIdx.x;
    int t_len = min(1024, length - blockIdx.x * blockDim.x);
    
    while(idx < length){
        // copy to shared mem
        sh_sums[threadIdx.x] = sums[idx];
        sh_ind[threadIdx.x] = ind[idx];

        __syncthreads();
        
        if(id%r == 0)
            dev_merge(sh_sums, sh_res, sh_ind, sh_indres, id, min(t_len, id + stride), min(t_len, id+r));
        
        __syncthreads();
        // copy result to global mem
        res[idx] = sh_res[threadIdx.x];
        ind_res[idx] = sh_indres[threadIdx.x];
        __syncthreads();
        idx += numthrds;
    }
}

// doesn't work
__global__
void kernal_merge_chuncks(float * sums, float * res, int * ind, int * ind_res, const unsigned int length, const unsigned int r){

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numthrds = blockDim.x * gridDim.x;

    const unsigned int stride = r/2;

    unsigned int id;
    while(idx*r < length){
        id = idx*r;
        dev_merge(sums, res, ind, ind_res, id, min(length, id + stride), min(length, id+r));
        idx += numthrds;
    }
}

__global__ 
void kernal_update_faces(vertex * f_in, vertex * f_out, int * inds, const unsigned int vertices_length){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numthrds = blockDim.x * gridDim.x;

    while(idx < vertices_length){
        f_out[idx] = f_in[inds[idx]];
        idx += numthrds;
    }
}

void cudacall_sort(int thread_num) {
    
    unsigned int len = 3*faces_length;
    int n_blocks = min(65535, (len + thread_num  - 1) / thread_num);
    
    unsigned int l = ceil(log2(len)), ind1;

    for(int i=0; i<l; i++){
        ind1 = i%2;
        ind2_sums = (i+1)%2;
        ind2_inds = ind2_sums;
        unsigned int r = pow(2, i+1);
        // kernal_merge_navie_sort<<<n_blocks, thread_num>>>(pointers_sums[ind1], pointers_sums[ind2_sums], pointers_inds[ind1], pointers_inds[ind2_inds], len, r);
        kernal_merge_sort<<<n_blocks, thread_num>>>(pointers_sums[ind1], pointers_sums[ind2_sums], pointers_inds[ind1], pointers_inds[ind2_inds], len, r);
        
    }    
    
    // now sort the chunks of 1024 floats
    l = ceil(log2(n_blocks));
    for(int i=0; i<l; i++){
        ind1 = (ind1+1)%2;
        ind2_sums = (ind2_sums+1)%2;
        ind2_inds = ind2_sums;
        unsigned int r = pow(2, i+1)*1024;
        kernal_merge_navie_sort<<<n_blocks, thread_num>>>(pointers_sums[ind1], pointers_sums[ind2_sums], pointers_inds[ind1], pointers_inds[ind2_inds], len, r);
    }
    
    // CUDA_CALL(hipMemcpy(sums, pointers_sums[ind2_sums], len*sizeof(float), hipMemcpyDeviceToHost));
    // CUDA_CALL(hipMemcpy(tmp, pointers_inds[ind2_inds], len*sizeof(int), hipMemcpyDeviceToHost));
    
    // working
    n_blocks = std::min(65535, ((int)len + thread_num  - 1) / thread_num);
    int out = (ind2_faces + 1) %2;
    kernal_update_faces<<<n_blocks, thread_num>>>((vertex *)pointers[ind2_faces], (vertex *)pointers[out], pointers_inds[ind2_inds], len);
    hipDeviceSynchronize();
    ind2_faces = out;
}


__device__ void gpu_spherical_harmonics(float radius, const int n_sph, vertex dev_R_vec, float* dev_coeff, float* U, int thread_index){

    float dev_V[21*21];
    float dev_W[21*21];

    // Define pseudo coefficients
    float Radius_sq = powf(radius,2);
    float rho = powf(R_eq,2)/Radius_sq;

    float x0 = R_eq*dev_R_vec.x/Radius_sq;
    float y0 = R_eq*dev_R_vec.y/Radius_sq;
    float z0 = R_eq*dev_R_vec.z/Radius_sq;

    // Calculate zonal terms V(n, 0). Set W(n,0)=0.0
    dev_V[0]= R_eq /sqrtf(Radius_sq);
    dev_W[0] = 0.0;

    dev_V[1*(n_sph+1) + 0] = z0 *dev_V[0];
    dev_W[1*(n_sph+1) + 0] = 0.0;

    for (int n=2; n<n_sph+1; n++){
        dev_V[n*(n_sph+1) + 0] = ((2*n-1)*z0*dev_V[(n-1)*(n_sph+1) + 0] - (n-1)*rho*dev_V[(n-2)*(n_sph+1) + 0])/n;
        dev_W[n*(n_sph+1) + 0] = 0.0;
    } // Eqn 3.30


    //Calculate tesseral and sectoral terms
    for (int m = 1; m < n_sph + 1; m++){
        // Eqn 3.29
        dev_V[m*(n_sph+1) + m] = (2*m-1)*(x0*dev_V[(m-1)*(n_sph+1) + (m-1)] - y0*dev_W[(m-1)*(n_sph+1) + (m-1)]);
        dev_W[m*(n_sph+1) + m] = (2*m-1)*(x0*dev_W[(m-1)*(n_sph+1) + (m-1)] + y0*dev_V[(m-1)*(n_sph+1) + (m-1)]);

        // n=m+1 (only one term)
        if (m < n_sph){
            dev_V[(m+1)*(n_sph+1) + (m)] = (2*m+1)*z0*dev_V[m*(n_sph+1) + m];
            dev_W[(m+1)*(n_sph+1) + (m)] = (2*m+1)*z0*dev_W[m*(n_sph+1) + m] ;
        }

        for (int n = m+2; n<n_sph+1; n++){
            dev_V[n*(n_sph+1) + m] = ((2*n-1)*z0*dev_V[(n-1)*(n_sph+1) + m]-(n+m-1)*rho*dev_V[(n-2)*(n_sph+1) + m])/(n-m);
            dev_W[n*(n_sph+1) + m] = ((2*n-1)*z0*dev_W[(n-1)*(n_sph+1) + m]-(n+m-1)*rho*dev_W[(n-2)*(n_sph+1) + m])/(n-m);
        }
    }

    // Calculate potential
    float C = 0; // Cnm coeff
    float S = 0; // Snm coeff
    float N = 0; // normalisation number
    float p = 1.0;
    U[thread_index] = 0; //potential
    for (int m=0; m<n_sph+1; m++){
        for (int n = m; n<n_sph+1; n++){
            C = 0;
            S = 0;
            if (m==0){
                N = sqrtf(2*n+1);
                C = N*dev_coeff[n*(n_sph+2)+0];
//                U[thread_index] = C*dev_V[n*(n_sph+1) + 0];
            }
            else {
                p = 1.0;
                // gpu_facprod(n,m,&p);
                for (int i = n-m+1; i<=n+m; i++){
                    p = p/i;
                }
                N = sqrtf((2)*(2*n+1)*p);
                C = N*dev_coeff[n*(n_sph+2)+m];
                S = N*dev_coeff[(n_sph-n)*(n_sph+2)+ (n_sph-m+1)];
            }
            U[thread_index] = U[thread_index] + C*dev_V[n*(n_sph+1) + m] + S*dev_W[n*(n_sph+1) + m];
            // Calculation of the Gravitational Potential Calculation model
        }
    }
    U[thread_index] = U[thread_index]*mhu/R_eq;
}


__global__
void naive_kernel_gravitational(int g_vertices_length, float g_radius, const int n_sph, float* dev_coeff, vertex* dev_vertices, float* dev_potential){


    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    while (thread_index < g_vertices_length) {

        gpu_spherical_harmonics(g_radius, n_sph, dev_vertices[thread_index], dev_coeff, dev_potential, thread_index);
        thread_index += blockDim.x * gridDim.x;
    }

}

__global__
void optimal_kernel_gravitational(int g_vertices_length, float radius, float eq_R, const int n_sph, float* dev_coeff, vertex* dev_vertices, float* U, int* M, int* N){


//    int thread_index = (blockIdx.x * blockDim.x + threadIdx.x);
//    int potential_index = (blockIdx.x * blockDim.x + threadIdx.x)%231;

    int potential_index = blockIdx.x;

    float dev_V[21*21];
    float dev_W[21*21];

    // Define pseudo coefficients
    float Radius_sq = powf(radius,2);
    float rho = powf(eq_R,2)/Radius_sq;

    float x0 = eq_R*dev_vertices[potential_index].x/Radius_sq;
    float y0 = eq_R*dev_vertices[potential_index].y/Radius_sq;
    float z0 = eq_R*dev_vertices[potential_index].z/Radius_sq;

    // Calculate zonal terms V(n, 0). Set W(n,0)=0.0
    dev_V[0]= eq_R/radius;
    dev_W[0] = 0.0;

    dev_V[1*(n_sph+1) + 0] = z0 *dev_V[0];
    dev_W[1*(n_sph+1) + 0] = 0.0;

    for (int n=2; n<n_sph+1; n++){
        dev_V[n*(n_sph+1) + 0] = ((2*n-1)*z0*dev_V[(n-1)*(n_sph+1) + 0] - (n-1)*rho*dev_V[(n-2)*(n_sph+1) + 0])/n;
        dev_W[n*(n_sph+1) + 0] = 0.0;
    } // Eqn 3.30

    //Calculate tesseral and sectoral terms
    for (int m = 1; m < n_sph + 1; m++){
        // Eqn 3.29
        dev_V[m*(n_sph+1) + m] = (2*m-1)*(x0*dev_V[(m-1)*(n_sph+1) + (m-1)] - y0*dev_W[(m-1)*(n_sph+1) + (m-1)]);
        dev_W[m*(n_sph+1) + m] = (2*m-1)*(x0*dev_W[(m-1)*(n_sph+1) + (m-1)] + y0*dev_V[(m-1)*(n_sph+1) + (m-1)]);

        // n=m+1 (only one term)
        if (m < n_sph){
            dev_V[(m+1)*(n_sph+1) + (m)] = (2*m+1)*z0*dev_V[m*(n_sph+1) + m];
            dev_W[(m+1)*(n_sph+1) + (m)] = (2*m+1)*z0*dev_W[m*(n_sph+1) + m] ;
        }

        for (int n = m+2; n<n_sph+1; n++){
            dev_V[n*(n_sph+1) + m] = ((2*n-1)*z0*dev_V[(n-1)*(n_sph+1) + m]-(n+m-1)*rho*dev_V[(n-2)*(n_sph+1) + m])/(n-m);
            dev_W[n*(n_sph+1) + m] = ((2*n-1)*z0*dev_W[(n-1)*(n_sph+1) + m]-(n+m-1)*rho*dev_W[(n-2)*(n_sph+1) + m])/(n-m);
        }
    }

        // thread index for the block and shared memory
        unsigned int tid = threadIdx.x;

        __shared__ float shmem[256]; //stores CV+SW
        shmem[tid] = 0.0; //potential

        // Calculate potential
        float C = 0; // Cnm coeff
        float S = 0; // Snm coeff
        float Norm = 0; // normalisation number
        float p = 1.0;

        if (tid<N_coeff){
            int n = N[tid];
            int m = M[tid];

            if (m==0){
                    Norm = sqrtf(2*n+1);
                    C = Norm*dev_coeff[n*(n_sph+2)+0];
                    shmem[tid] = C*dev_V[n*(n_sph+1) + 0];
                }
                else {
                    p = 1.0;
                    for (int i = n-m+1; i<=n+m; i++){
                        p = p/i;
                    }
                    Norm = sqrtf((2)*(2*n+1)*p);
                    C = Norm*dev_coeff[n*(n_sph+2)+m];
                    S = Norm*dev_coeff[(n_sph-n)*(n_sph+2)+ (n_sph-m+1)];
                    shmem[tid] = C*dev_V[n*(n_sph+1) + m] + S*dev_W[n*(n_sph+1) + m];
                }
        }
        // Calculation of the Gravitational Potential Calculation model

        // sync threads before commencing the stages of reduction
        __syncthreads();

        // Reduction #3: Sequential Addressing
        // Ref: Presentation "Optimizing Parallel Reduction in CUDA", by Mark Harris.
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            // conduct the summation
            shmem[tid] = shmem[tid] + shmem[tid + s];
//                atomicAdd(&shmem[tid], shmem[tid + s]);
        }
        // Sync threads after every stage of reduction
        __syncthreads();
    }

//    U[potential_index] = shmem[0]*mhu/R_eq;
    U[potential_index] = shmem[0];
//        thread_index += blockDim.x * gridDim.x;
//    }

}

void optimal_cudacall_gravitational(int thread_num){

//    int n_blocks = ceil(vertices_length*1.0/thread_num);
//    n_blocks = std::min(65535, n_blocks);

    int len = vertices_length;
//    int n_blocks = std::min(65535, (len + thread_num  - 1) / thread_num);
    int n_blocks = std::min(65535, len);
    cout<<"\n Number of blocks \t"<<n_blocks<<'\n';

    int M[N_coeff];
    int N[N_coeff];

    int k = 0;
    for (int n=0;n<N_SPHERICAL+1;n++){
        for (int m=0;m<n+1;m++){
            N[k] = n;
            M[k] = m;
            k++;
        }
    }

    int* dev_M;
    int* dev_N;

    CUDA_CALL(hipMalloc((void**) &dev_N, sizeof(int) * N_coeff));
    CUDA_CALL(hipMalloc((void**) &dev_M, sizeof(int) * N_coeff));
    CUDA_CALL(hipMemcpy(dev_N, N, sizeof(int) * N_coeff, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(dev_M, M, sizeof(int) * N_coeff, hipMemcpyHostToDevice));
    optimal_kernel_gravitational<<<n_blocks, thread_num>>>(vertices_length, radius, R_eq, N_SPHERICAL, dev_coeff, dev_vertices, dev_potential, dev_M, dev_N);
    CUDA_CALL(hipFree(dev_M));
    CUDA_CALL(hipFree(dev_N));
}





void naive_cudacall_gravitational(int thread_num){

//    int n_blocks = ceil(vertices_length*1.0/thread_num);
//    n_blocks = std::min(65535, n_blocks);

    int len = vertices_length;
    int n_blocks = std::min(65535, (len + thread_num  - 1) / thread_num);

    naive_kernel_gravitational<<<n_blocks, thread_num>>>(vertices_length, radius, N_SPHERICAL, dev_coeff, dev_vertices, dev_potential);
}

