#include "hip/hip_runtime.h"
/* 
 * CUDA blur
 */
#ifndef _GRAV_CUDA_C_
	#define _GRAV_CUDA_C_
#endif

#include "grav_cuda.cuh"
#include ""
#include "cuda_header.cuh"

#include "grav_cpu.h"
#include "cuda_calls_helper.h"

#include <cstdio>
#include <cstdlib>
#include <iostream>
using std::cerr;
using std::cout;
using std::endl;


CUDA_CALLABLE
void break_triangle(triangle face_tmp, vertex * v_tmp, float radius) {
	float x_tmp, y_tmp, z_tmp, scale;
    for(int i=0; i<3; i++){
    	x_tmp = (face_tmp.v[i].x + face_tmp.v[(i+1)%3].x)/2;
		y_tmp = (face_tmp.v[i].y + face_tmp.v[(i+1)%3].y)/2;
		z_tmp = (face_tmp.v[i].z + face_tmp.v[(i+1)%3].z)/2;
		scale = radius/sqrtf(x_tmp*x_tmp + y_tmp*y_tmp + z_tmp*z_tmp);
		v_tmp[i].x = x_tmp*scale;
		v_tmp[i].y = y_tmp*scale;
		v_tmp[i].z = z_tmp*scale;
    }
}

__global__ void refine_icosphere_naive_kernal(triangle * faces, float radius, unsigned int depth) {

	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int numthrds = blockDim.x * gridDim.x;

	unsigned int th_len, write_offset;

	vertex v_tmp[3];
		
	th_len = 20*pow(4, depth);
	while(idx < th_len){

		triangle tri_tmp = faces[idx];
		write_offset = th_len + 3*idx;
		
		break_triangle(tri_tmp, v_tmp, radius);
		// got the mid points of the vertices now make new triangles
		faces[idx].v[1] = v_tmp[0];
		faces[idx].v[2] = v_tmp[2];

		// adding triangle V[0], P1, V[1]
		faces[write_offset].v[0] = v_tmp[0];
		faces[write_offset].v[1] = tri_tmp.v[1];
		faces[write_offset].v[2] = v_tmp[1];
		write_offset++;

		//adding triangle P2, V[1], V[2]
		faces[write_offset].v[0] = v_tmp[1];
		faces[write_offset].v[1] = tri_tmp.v[2];
		faces[write_offset].v[2] = v_tmp[2];
		write_offset++;

		//adding triangle V[0], V[1], V[2]
		faces[write_offset].v[0] = v_tmp[0];
		faces[write_offset].v[1] = v_tmp[1];
		faces[write_offset].v[2] = v_tmp[2];
		write_offset++;
		
		idx += numthrds;
	}
    
}

void cudacall_icosphere_naive(int thread_num) {

	// each thread works on one face
	for(int i=0; i<max_depth; i++){
		int ths = 20*pow(4, i);
		int n_blocks = std::min(65535, (ths + thread_num  - 1) / thread_num);
		refine_icosphere_naive_kernal<<<n_blocks, thread_num>>>(dev_faces, radius, i);
	}
	
}

__global__ void refine_icosphere_sh_naive_kernal(triangle * faces, float radius, unsigned int depth) {

	extern __shared__ triangle sh_faces[];
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	unsigned int th_len, write_offset;

	vertex v_tmp[3];

	th_len = 20*pow(4, depth);
	const int id = 4*threadIdx.x;
	// copy from faces to sh_faces
	if(idx < th_len)
		sh_faces[id] = faces[idx];
	__syncthreads();
		
	if(idx < th_len){

		triangle tri_tmp = sh_faces[id];
		
		break_triangle(tri_tmp, v_tmp, radius);
		// got the mid points of the vertices now make new triangles
		sh_faces[id].v[1] = v_tmp[0];
		sh_faces[id].v[2] = v_tmp[2];

		// adding triangle V[0], P1, V[1]
		sh_faces[id+1].v[0] = v_tmp[0];
		sh_faces[id+1].v[1] = tri_tmp.v[1];
		sh_faces[id+1].v[2] = v_tmp[1];

		//adding triangle P2, V[1], V[2]
		sh_faces[id+2].v[0] = v_tmp[1];
		sh_faces[id+2].v[1] = tri_tmp.v[2];
		sh_faces[id+2].v[2] = v_tmp[2];

		//adding triangle V[0], V[1], V[2]
		sh_faces[id+3].v[0] = v_tmp[0];
		sh_faces[id+3].v[1] = v_tmp[1];
		sh_faces[id+3].v[2] = v_tmp[2];
		
		// idx += numthrds;
	}
	// copy to global memory now
	write_offset = th_len + 3*idx;

	faces[idx] = sh_faces[id];
	faces[write_offset] = sh_faces[id+1];
	faces[write_offset+1] = sh_faces[id+2];
	faces[write_offset+2] = sh_faces[id+3];
    
}

void cudacall_icosphere_sh_naive(int thread_num) {
	// each thread works on one face
	for(int i=0; i<max_depth; i++){
		int ths = 20*pow(4, i);
		int n_blocks = std::min(65535, (ths + thread_num  - 1) / thread_num);
		refine_icosphere_sh_naive_kernal<<<n_blocks, thread_num, 4*thread_num*sizeof(triangle)>>>(dev_faces, radius, i);
	}
	
}

__global__ void refine_icosphere_kernal(triangle * faces, float radius, unsigned int depth) {

	// TODO implement the shared memory

	// extern __shared__ float shmem[];

	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int numthrds = blockDim.x * gridDim.x;
    // unsigned tid = threadIdx.x;

	unsigned int depth_c, th_len, write_offset;

	vertex v_tmp[3];
	vertex v_storage[12];
	depth_c  = depth;
		
	th_len = 20*pow(4, depth_c);
	while(idx < 4*th_len){
		int tri_ind = idx/4;
		int sub_tri_ind = (int)idx%4;

		triangle tri_tmp = faces[tri_ind];
		write_offset = (sub_tri_ind == 0)*tri_ind + (sub_tri_ind!=0)*(th_len + 3*tri_ind + (idx%4-1));
		break_triangle(tri_tmp, v_tmp, radius);

		v_storage[0] = tri_tmp.v[0];
		v_storage[1] = v_tmp[0];
		v_storage[2] = v_tmp[2];
		
		v_storage[3] = v_tmp[0];
		v_storage[4] = tri_tmp.v[1];
		v_storage[5] = v_tmp[1];

		v_storage[6] = v_tmp[1];
		v_storage[7] = tri_tmp.v[2];
		v_storage[8] = v_tmp[2];
		
		v_storage[9] = v_tmp[0];
		v_storage[10] = v_tmp[1];
		v_storage[11] = v_tmp[2];

	
		faces[write_offset].v[0] = v_storage[3*sub_tri_ind];
		faces[write_offset].v[1] = v_storage[3*sub_tri_ind+1];
		faces[write_offset].v[2] = v_storage[3*sub_tri_ind+2];
	
		idx += numthrds;
	}

}
void cudacall_icosphere(int thread_num) {
	// each thread works on one face
	for(int i=0; i<max_depth; i++){
		int ths = 20*pow(4, i);

		// int * dev_res, * gpu_res_out;
		// CUDA_CALL(hipMalloc((void **)&dev_res, 4*ths * sizeof(int)));
		// gpu_res_out = (int *)malloc(4*ths*sizeof(int));
		thread_num = thread_num - thread_num%4;
		int n_blocks = std::min(65535, (4*ths + thread_num  - 1) / thread_num);
		refine_icosphere_kernal<<<n_blocks, thread_num>>>(dev_faces, radius, i);

		// CUDA_CALL(hipMemcpy(gpu_res_out, dev_res, 4*ths*sizeof(int), hipMemcpyDeviceToHost));
		// CUDA_CALL(hipFree(dev_res));
		// for(int j=0; j<4*ths; j++){
		// 	printf("j: %d is_not_zero %d, \n", j, gpu_res_out[j]);
		// }
	}

}

void cuda_cpy_input_data(){
	gpu_out_faces = (triangle *)malloc(faces_length*sizeof(triangle));
	CUDA_CALL(hipMalloc((void **)&dev_faces, faces_length * sizeof(triangle)));
	CUDA_CALL(hipMemcpy(dev_faces, faces_init, ICOSPHERE_INIT_FACE_LEN*sizeof(triangle), hipMemcpyHostToDevice));
}

void cuda_cpy_output_data(){
	CUDA_CALL(hipMemcpy(gpu_out_faces, dev_faces, faces_length*sizeof(triangle), hipMemcpyDeviceToHost));
}

void free_gpu_memory(){
	CUDA_CALL(hipFree(dev_faces));
	free(gpu_out_faces);
}