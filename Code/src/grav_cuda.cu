#include "hip/hip_runtime.h"
/*
 * CUDA blur
 */
#ifndef _GRAV_CUDA_C_
	#define _GRAV_CUDA_C_
#endif

#include "grav_cuda.cuh"
#include ""
#include "cuda_header.cuh"

#include "grav_cpu.hpp"
#include "cuda_calls_helper.h"

#include <cstdio>
#include <cstdlib>
#include <iostream>
using std::cerr;
using std::cout;
using std::endl;




__device__ void break_triangle(triangle face_tmp, vertex * v_tmp, float radius) {
	float x_tmp, y_tmp, z_tmp, scale;
    for(int i=0; i<3; i++){
    	x_tmp = (face_tmp.v[i].x + face_tmp.v[(i+1)%3].x)/2;
		y_tmp = (face_tmp.v[i].y + face_tmp.v[(i+1)%3].y)/2;
		z_tmp = (face_tmp.v[i].z + face_tmp.v[(i+1)%3].z)/2;
		scale = radius/sqrtf(x_tmp*x_tmp + y_tmp*y_tmp + z_tmp*z_tmp);
		v_tmp[i].x = x_tmp*scale;
		v_tmp[i].y = y_tmp*scale;
		v_tmp[i].z = z_tmp*scale;
    }
}

__global__ void refine_icosphere_naive_kernal(triangle * faces, const float radius, const unsigned int depth) {

	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int numthrds = blockDim.x * gridDim.x;

	unsigned int  write_offset;

	vertex v_tmp[3];

	const unsigned int th_len = 20*pow(4, depth);
	while(idx < th_len){

		triangle tri_tmp = faces[idx];
		write_offset = th_len + 3*idx;

		break_triangle(tri_tmp, v_tmp, radius);
		// got the mid points of the vertices now make new triangles
		faces[idx].v[1] = v_tmp[0];
		faces[idx].v[2] = v_tmp[2];

		// adding triangle V[0], P1, V[1]
		faces[write_offset].v[0] = v_tmp[0];
		faces[write_offset].v[1] = tri_tmp.v[1];
		faces[write_offset].v[2] = v_tmp[1];
		write_offset++;

		//adding triangle P2, V[1], V[2]
		faces[write_offset].v[0] = v_tmp[1];
		faces[write_offset].v[1] = tri_tmp.v[2];
		faces[write_offset].v[2] = v_tmp[2];
		write_offset++;

		//adding triangle V[0], V[1], V[2]
		faces[write_offset].v[0] = v_tmp[0];
		faces[write_offset].v[1] = v_tmp[1];
		faces[write_offset].v[2] = v_tmp[2];
		write_offset++;

		idx += numthrds;
	}

}

void cudacall_icosphere_naive(int thread_num) {
	// each thread works on one face
	for(int i=0; i<max_depth; i++){
		int ths = 20*pow(4, i);
		int n_blocks = std::min(65535, (ths + thread_num  - 1) / thread_num);
		refine_icosphere_naive_kernal<<<n_blocks, thread_num>>>(dev_faces_in, radius, i);
	}

}


typedef void (*func_ptr_sub_triangle_t)(triangle, vertex *, triangle *);

__device__ void sub_triangle_top(triangle face_tmp, vertex * v_tmp, triangle * res) {
    res->v[0] = face_tmp.v[0];
    res->v[1] = v_tmp[0];
    res->v[2] = v_tmp[2];
}

__device__ void sub_triangle_left(triangle face_tmp, vertex * v_tmp, triangle * res) {
    res->v[0] = v_tmp[0];
    res->v[1] = face_tmp.v[1];
    res->v[2] = v_tmp[1];
}

__device__ void sub_triangle_right(triangle face_tmp, vertex * v_tmp, triangle * res) {
    res->v[0] = v_tmp[1];
    res->v[1] = face_tmp.v[2];
    res->v[2] = v_tmp[2];
}

__device__ void sub_triangle_center(triangle face_tmp, vertex * v_tmp, triangle * res) {
    res->v[0] = v_tmp[0];
    res->v[1] = v_tmp[1];
    res->v[2] = v_tmp[2];
}

__device__ func_ptr_sub_triangle_t funcs2[4] = {sub_triangle_top, sub_triangle_left, sub_triangle_right, sub_triangle_center};


__global__ void refine_icosphere_kernal(triangle * faces, const float radius, const unsigned int th_len, triangle * faces_out) {

	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int numthrds = blockDim.x * gridDim.x;

	vertex v_tmp[3];

	while(idx < 4*th_len){
		int tri_ind = idx/4;
		int sub_tri_ind = idx%4;

		break_triangle(faces[tri_ind], v_tmp, radius);

		funcs2[sub_tri_ind](faces[tri_ind], v_tmp, &faces_out[idx]);

		idx += numthrds;
	}
}

// variables local to this file
int ind2;
triangle* pointers[2];
void cudacall_icosphere(int thread_num) {

	// each thread creates a sub triangle
	int ths, n_blocks, ind1;
	for(int i=0; i<max_depth; i++){
		ths = 20*pow(4, i);
		n_blocks = std::min(65535, (ths + thread_num  - 1) / thread_num);
		ind1 = i%2;
		ind2 = (i+1)%2;
		refine_icosphere_kernal<<<n_blocks, thread_num>>>(pointers[ind1], radius, ths, pointers[ind2]);
	}
}

void cuda_cpy_input_data(){
	gpu_out_faces = (triangle *)malloc(faces_length*sizeof(triangle));
	CUDA_CALL(hipMalloc((void **)&dev_faces_in, faces_length * sizeof(triangle)));
	CUDA_CALL(hipMalloc((void **)&dev_faces_out, faces_length * sizeof(triangle)));
	CUDA_CALL(hipMemcpy(dev_faces_in, faces_init, ICOSPHERE_INIT_FACE_LEN*sizeof(triangle), hipMemcpyHostToDevice));

	pointers[0] = dev_faces_in;
	pointers[1] = dev_faces_out;
}

void cuda_cpy_output_data(){
	CUDA_CALL(hipMemcpy(gpu_out_faces, pointers[ind2], faces_length*sizeof(triangle), hipMemcpyDeviceToHost));
}

void free_gpu_memory(){
	CUDA_CALL(hipFree(dev_faces_in));
	CUDA_CALL(hipFree(dev_faces_out));
	free(gpu_out_faces);
}
